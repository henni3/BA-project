#include "tsp-main-helper.cu.h"

int main(int argc, char* argv[]) {
    if (argc != 3) {
        printf("Usage: %s <file-name> <number-of-restarts (multiple of 50)> \n", argv[0]);
        exit(1);
    }
    // Collect input arguments
    char* file_name = argv[1];
    int restarts = atoi(argv[2]);
    if (restarts % 10 != 0) {
        printf("restarts must be multiple of 10 \n");
        exit(1);
    } 
    if(restarts <= 0){
        printf("Number of restarts has to be a number larger than 0");
        exit(1);
    }
    initHwd();
    for (int i = 50; i <= restarts; i += 50) {
        runProgram(file_name, i, 4);
        //printf("iteration %d \n", i);
    }

    return 0;
}