#include "tsp-main-helper.cu.h"

int main(int argc, char* argv[]) {
    if (argc != 4) {
        printf("Usage: %s <block-size> <file-name> <number-of-restarts>\n", argv[0]);
        exit(1);
    }
    // Collect input arguments
    int block_size = atoi(argv[1]);
    char* file_name = argv[2];
    int restarts = atoi(argv[3]);
    if(restarts <= 0){
        printf("Number of restarts has to be a number larger than 0");
        exit(1);
    }
    
    initHwd();

    //Create varibales
    struct timeval randomTime, start, end, diff;
    uint32_t* distMatrix, *kerDist, num_blocks_tour, num_blocks_gl_re;
    int cities, totIter, *is_d, *js_d, *glo_results, *glo_res_h, tourId, REPEAT, elapsed;
    unsigned short *tourMatrixIn_d, *tourMatrixTrans_d, *tourMatrix_h;
    size_t mult_sharedMem;


    // Collect information from datafile into distMatrix and cities    
    distMatrix = (uint32_t*) malloc(sizeof(uint32_t) * MAXCITIES * MAXCITIES);
    cities = fileToDistM(file_name, distMatrix);
    if( cities > MAXCITIES){
        printf("too many cities :( \n");
        exit(1);
    }
    distMatrix = (uint32_t*) realloc(distMatrix,sizeof(uint32_t)* cities * cities);
    hipMalloc((void**)&kerDist, cities*cities*sizeof(uint32_t));
    hipMemcpy(kerDist, distMatrix, cities*cities*sizeof(uint32_t), hipMemcpyHostToDevice);

    //Calculate total number of iterations
    totIter = ((cities-1) * (cities-2))/2;

    //Cuda malloc
    hipMalloc((void**)&tourMatrixIn_d, (cities+1)*restarts*sizeof(unsigned short));
    hipMalloc((void**)&tourMatrixTrans_d, (cities+1)*restarts*sizeof(unsigned short));
    hipMalloc((void**)&is_d, totIter*sizeof(uint32_t));
    hipMalloc((void**)&js_d, totIter*sizeof(uint32_t));
    hipMalloc((void**)&glo_results, 2*restarts*sizeof(int));
    int* restart_array;
    hipMalloc((void**)&restart_array, restarts * sizeof(int));

    //CPU malloc
    glo_res_h = (int*) malloc(2*restarts*sizeof(int));
    tourMatrix_h = (unsigned short*) malloc((cities+1)*restarts*sizeof(unsigned short));

    //testing timer for cities 100 program
    REPEAT = 0;
    gettimeofday(&start, NULL); 
    while(REPEAT < 10){
        init(block_size, cities, totIter, is_d, js_d);

        //Prepare for column wise tour
        num_blocks_tour = (restarts + block_size-1)/block_size; 
        gettimeofday(&randomTime, NULL);
        int time = randomTime.tv_usec * 10;
        //Create tour matrix column wise
        createToursColumnWise<<<num_blocks_tour, block_size>>> (tourMatrixIn_d, cities, restarts, time);
        transposeTiled<unsigned short, TILE>(tourMatrixIn_d, tourMatrixTrans_d, (cities+1), restarts);
        //printf("size of change tuple = %d \n", sizeof(ChangeTuple));
        //run 2 opt kernel 
        size_t sharedMemSize = (cities+1) * sizeof(unsigned short) + block_size * sizeof(ChangeTuple) + sizeof(ChangeTuple);
        //printf("sharedmemSize used in twoOptKer : %d \n", sharedMemSize);

        twoOptKer<<<restarts, block_size, sharedMemSize>>> (kerDist, tourMatrixTrans_d, 
                                                        is_d, glo_results, 
                                                        cities, totIter, restart_array);
        //run reduction of all local optimum cost across multiple blocks
        num_blocks_gl_re = (num_blocks_tour+1)/2;
        mult_sharedMem = (block_size*2) * sizeof(int);
        for(int i = num_blocks_gl_re; i > 1; i>>=1){
            multBlockReduce<<<i, block_size, mult_sharedMem>>>(glo_results, restarts);
            i++;
        }
        //run reduction on the last block
        multBlockReduce<<<1, block_size, mult_sharedMem>>>(glo_results, restarts);

        //print results
        hipMemcpy(glo_res_h, glo_results, 2*restarts*sizeof(int), hipMemcpyDeviceToHost);
        int* host_restart = (int*) malloc(restarts * sizeof(int));
        hipMemcpy(host_restart, restart_array, restarts* sizeof(int),hipMemcpyDeviceToHost);
        int re_sum = 0;
        for (int i = 0; i < restarts; i++){
            re_sum += host_restart[i];
        }
        float average = (float) re_sum /  (float) restarts;
        printf("average nr. of restarts is %f, for %d climbers \n", average, restarts);
        hipFree(restart_array);
        free(host_restart);

        
        //tour matrix row wise
        hipMemcpy(tourMatrix_h, tourMatrixTrans_d, (cities+1)*restarts*sizeof(unsigned short), hipMemcpyDeviceToHost);
        
        tourId = glo_res_h[1];
        REPEAT++;
    }
    hipDeviceSynchronize();
    gettimeofday(&end, NULL); 
    timeval_subtract(&diff, &end, &start);
    elapsed = (diff.tv_sec*1e6+diff.tv_usec) / REPEAT; 
    printf("Original kernel: Optimized Program runs on GPU in: %lu milisecs, repeats: %d\n", elapsed/1000, REPEAT);
    
    printf("Shortest path: %d\n", glo_res_h[0]);
    printf("Tour:  [");
    for(int i = 0; i < cities+1; i++){
        printf("%d, ", tourMatrix_h[(cities+1)*tourId+i]);
    }
    printf("]\n");
    
    hipFree(tourMatrixIn_d);
    free(distMatrix); free(tourMatrix_h); free(glo_res_h); 
    hipFree(is_d); hipFree(js_d); hipFree(tourMatrixTrans_d); 
    hipFree(kerDist);
    hipFree(glo_results);
    return 0;
}