#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hostSkel.cu.h"
#include "tsp-kernels.cu.h"

int init(int block_size, 
         int cities, 
         int totIter, 
         int* is_d, 
         int* js_d){
    int len, *index_shp_d, *index_shp_sc_d, *d_tmp_int;
    int *flag_int, *oneArr, *seg_sc_tmp_int;
    char *flags_d, *d_tmp_flag; 
    
    //Calculate the length of shape array
    len = cities - 2;
    //Calculate block size
    unsigned int num_blocks     = (totIter + block_size-1)/block_size; 
    unsigned int num_blocks_shp = (len + block_size-1)/block_size; 

    //Cuda malloc 
    hipMalloc((void**)&index_shp_d,    len*sizeof(int));
    hipMalloc((void**)&index_shp_sc_d, len*sizeof(int));
    hipMalloc((void**)&flags_d,        totIter*sizeof(char));
    hipMalloc((void**)&oneArr,         totIter*sizeof(int));

    hipMalloc((void**)&d_tmp_int,      MAX_BLOCK*sizeof(int));
    hipMalloc((void**)&seg_sc_tmp_int,   MAX_BLOCK*sizeof(int));
    hipMalloc((void**)&d_tmp_flag,     totIter*sizeof(char));
    //Create shape array for index
    mkIndShp<<< num_blocks, block_size >>> (index_shp_d, len);

    // Make flag array
    // 1. scan the shape array
    scanInc<Add<int> > (block_size, len, index_shp_sc_d, index_shp_d, d_tmp_int);

    // 2. create an array of zeros
    replicate0<<< num_blocks, block_size >>> (totIter, flags_d);
    
    // 3. scatter the flag array
    mkFlags<<< num_blocks_shp, block_size >>> (totIter, index_shp_sc_d, flags_d);
    hipMalloc((void**)&flag_int,       totIter*sizeof(int));
    convert<<< num_blocks_shp, block_size >>> (flag_int, flags_d, totIter);
    
    /*
    int* flag = (int*) malloc(totIter*sizeof(int));
    hipMemcpy(flag, flag_int, totIter*sizeof(int), hipMemcpyDeviceToHost);
    printf("flag: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", flag[i]);
    }
    printf("]\n");
    free(flag); */

    /*
    char* flag_c = (char*) malloc(totIter*sizeof(char));
    hipMemcpy(flag_c, flags_d, totIter*sizeof(char), hipMemcpyDeviceToHost);
    printf("flag_c: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", flag_c[i]);
    }
    printf("]\n");
    free(flag_c); */


    //Make is array
    // 1. scan the flag array
    scanInc<Add<int> > (block_size, totIter, is_d, flag_int, d_tmp_int);
    // 2. minus each element of is_d array with one to get the final is_d array
     minusOne<<< num_blocks, block_size >>> (totIter, is_d);
 
    //Make js array
    // 1. create an array of ones
    replicate1<<< num_blocks, block_size >>> (totIter, oneArr);
    // 2. segmented scan on the flag array
    sgmScanInc<Add<int> > (block_size, totIter, js_d, flags_d, oneArr, seg_sc_tmp_int, d_tmp_flag);
    // 3. minus each element of js_d array with one to get the final js_d array
    minusOne<<< num_blocks, block_size >>> (totIter, js_d);

    
    //free cuda memory
    hipFree(index_shp_d);  hipFree(index_shp_sc_d);
    hipFree(flags_d);  hipFree(d_tmp_int);  hipFree(d_tmp_flag);
    return 0;
}


int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s <block-size>\n", argv[0]);
        exit(1);
    }
    
    initHwd();
    int cities = 5;
    //Calculate total number of iterations
    int totIter = ((cities-1) * (cities-2))/2;

    int block_size = atoi(argv[1]);
   // uint32_t totDist = cities * cities;
    //uint32_t* distM = (uint32_t*) malloc((totDist)*sizeof(uint32_t));
    //uint32_t* tour = (uint32_t*) malloc((cities + 1 ) * sizeof(uint32_t));
    //uint32_t tempDist[25] = {0,4,6,8,3,4,0,4,5,2,6,4,0,2,3,8,5,2,0,4,3,2,3,4,0};
    //uint32_t tempTour[6] = {4,2,0,3,1,4};

    //memcpy(distM, tempDist, sizeof(uint32_t) * (totDist));
    //memcpy(tour, tempTour, sizeof(uint32_t) * (cities+1));
    
    //Memory for i-array and j-array
    int *is_d, *js_d;
    hipMalloc((void**)&is_d, totIter*sizeof(uint32_t));
    hipMalloc((void**)&js_d, totIter*sizeof(uint32_t));


    init(block_size, cities, totIter, is_d, js_d);
    int* is_h = (int*) malloc(totIter*sizeof(uint32_t));
    hipMemcpy(is_h, is_d, totIter*sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("is: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", is_h[i]);
    }
    printf("]\n");

    int* js_h = (int*) malloc(totIter*sizeof(int));
    hipMemcpy(js_h, js_d, totIter*sizeof(int), hipMemcpyDeviceToHost);
    printf("js: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", js_h[i]);
    }
    printf("]\n");
    free(js_h); free(is_h);

    hipFree(is_d); hipFree(js_d);
    return 0;

    
}
