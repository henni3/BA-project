#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "hostSkel.cu.h"
#include "tsp-kernels.cu.h"

int twoOptMove(int block_size, int cities){
    int   totIter, *index_shp_d, *index_shp_sc_d, *d_tmp_int;
    char  *flags_d, *d_tmp_flag; 

    //Calculate total number of iterations
    totIter = ((cities-1) * (cities-2))/2;

    //Calculate the length of shape array
    len = cities - 2;

    //Calculate block size
    unsigned int num_blocks     = (totIter + block_size-1)/block_size; 
    unsigned int num_blocks_shp = (len + block_size-1)/block_size; 

    //Cuda malloc 
    hipMalloc((void**)&index_shp_d,    len*sizeof(int));
    hipMalloc((void**)&index_shp_sc_d, len*sizeof(int));
    hipMalloc((void**)&flags_d,        totIter*sizeof(char));

    hipMalloc((void**)&d_tmp_int,   MAX_BLOCK*sizeof(int));
    hipMalloc((void**)&d_tmp_flag,  MAX_BLOCK*sizeof(char));

    //Create shape array for index
    for(int i = 0; i < len; i++){
        index_shp_d[i] = len - i;
    }

    // Make flag array
    // 1. scan the shape array
    scanInc< Add<int> > (block_size, len, index_shp_sc_d, index_shp_d, d_tmp_int);

    // 2. create an array of zeros
    replicate0<<< num_blocks, block_size >>> ( totIter, flags_d );

    // 3. scatter the flag array
    mkFlags<<< num_blocks_shp, block_size >>> (len, index_shp_sc_d, flags_d);
    printf("index_shape_scan: [");
    for (int = 0; i < len; i++) {
        printf("%d, ", index_shp_sc_d[i]);
    }
    printf("] \n"); 

    printf("index_shape: [");
    for (int = 0; i < len; i++) {
        printf("%d, ", index_shp_d[i]);
    }
    printf("] \n");

    printf("flag_Arr: [");
    for (int = 0; i < len; i++) {
        printf("%d, ", flags_d [i]);
    }
    printf("] \n");
    //free cuda memory
    hipFree(index_shp_d);  hipFree(index_shp_sc_d);
    hipFree(flags_d);  hipFree(d_tmp_int);  hipFree(d_tmp_flag);
    return 0;
}


int main(int argc, char* argv[]) {
    int cities = 5;
    int block_size = atoi(argv[1]);
    uint32_t totDist = cities * cities;
    uint32_t* distM = (uint32_t*) malloc((totDist)*sizeof(uint32_t));
    uint32_t* tour = (uint32_t*) malloc((cities + 1 ) * sizeof(uint32_t));

    memcpy(distM, (uint32_t[25]){0,4,6,8,3,4,0,4,5,2,6,4,0,2,3,8,5,2,0,4,3,2,3,4,0}, sizeof(uint32_t) * (totDist));
    memcpy(tour, (int[6]) {4,2,0,3,1,4}, sizeof(int) * (cities+1));

    twoOptMove(block_size, cities);
    return 0;

    
}