#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "hostSkel.cu.h"
#include "tsp-kernels.cu.h"
#include "dataCollector.cu.h"

int init(int block_size, 
         int cities, 
         int totIter, 
         int* is_d, 
         int* js_d){
    int len, *index_shp_d, *index_shp_sc_d, *d_tmp_int;
    int *flag_int, *oneArr, *seg_sc_tmp_int;
    char *flags_d, *d_tmp_flag; 
    //Calculate the length of shape array
    len = cities - 2;
    //Calculate block size
    unsigned int num_blocks     = (totIter + block_size-1)/block_size; 
    unsigned int num_blocks_shp = (len + block_size-1)/block_size; 

    //Cuda malloc 
    hipMalloc((void**)&index_shp_d,    len*sizeof(int));
    hipMalloc((void**)&index_shp_sc_d, len*sizeof(int));
    hipMalloc((void**)&flags_d,        totIter*sizeof(char));
    hipMalloc((void**)&oneArr,         totIter*sizeof(int));

    hipMalloc((void**)&d_tmp_int,      MAX_BLOCK*sizeof(int));
    hipMalloc((void**)&seg_sc_tmp_int,   MAX_BLOCK*sizeof(int));
    hipMalloc((void**)&d_tmp_flag,     totIter*sizeof(char));
    //Create shape array for index
    mkIndShp<<< num_blocks, block_size >>> (index_shp_d, len);
    //hipDeviceSynchronize();
    /*int* indSha = (int*) malloc(len*sizeof(int));
    hipMemcpy(indSha, index_shp_d, len*sizeof(int), hipMemcpyDeviceToHost);
    printf("indSha: [");
    for(int i = 0; i < len; i++){
        printf("%d, ", indSha[i]);
    }
    printf("]\n \n");
    free(indSha);*/
    // Make flag array
    // 1. scan the shape array
    scanInc<Add<int> > (block_size, len, index_shp_sc_d, index_shp_d, d_tmp_int);
    //gpuErrchk( hipPeekAtLastError() );
  
    /*int* scan = (int*) malloc(len*sizeof(int));
    hipMemcpy(scan, index_shp_sc_d, len*sizeof(int), hipMemcpyDeviceToHost);
    printf("scan: [");
    for(int i = 0; i < len; i++){
        printf("%d, ", scan[i]);
    }
    printf("]\n \n");
    free(scan);*/
    // 2. create an array of zeros
    replicate0<<< num_blocks, block_size >>> (totIter, flags_d);
    
    // 3. scatter the flag array
    mkFlags<<< num_blocks_shp, block_size >>>(len, index_shp_sc_d, flags_d); // was totIter
    //gpuErrchk( hipPeekAtLastError() );
    //gpuErrchk( hipDeviceSynchronize() );
    hipMalloc((void**)&flag_int,       totIter*sizeof(int));
    convert<<< num_blocks, block_size >>> (flag_int, flags_d, totIter);
    
    
    /*int* flag = (int*) malloc(totIter*sizeof(int));
    hipMemcpy(flag, flag_int, totIter*sizeof(int), hipMemcpyDeviceToHost);
    printf("flag: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", flag[i]);
    }
    printf("]\n \n");
    free(flag);

    
    char* flag_c = (char*) malloc(totIter*sizeof(char));
    hipMemcpy(flag_c, flags_d, totIter*sizeof(char), hipMemcpyDeviceToHost);
    printf("flag_c: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", flag_c[i]);
    }
    printf("]\n");
    free(flag_c);*/

    //Make is array
    // 1. scan the flag array
    scanInc<Add<int> > (block_size, totIter, is_d, flag_int, d_tmp_int);
    // 2. minus each element of is_d array with one to get the final is_d array
    minusOne<<< num_blocks, block_size >>> (totIter, is_d);
 
    //Make js array
    // 1. create an array of ones
    replicate1<<< num_blocks, block_size >>> (totIter, oneArr);
    // 2. segmented scan on the flag array
    sgmScanInc<Add<int> > (block_size, totIter, js_d, flags_d, oneArr, seg_sc_tmp_int, d_tmp_flag);
    // 3. minus each element of js_d array with one to get the final js_d array
    minusOne<<< num_blocks, block_size >>> (totIter, js_d);
    zip<<< num_blocks, block_size>>> (is_d,js_d,totIter);


    hipDeviceSynchronize();

    
    //free cuda memory
    hipFree(index_shp_d);  hipFree(index_shp_sc_d);
    hipFree(flags_d);  hipFree(d_tmp_int);  hipFree(d_tmp_flag);
    return 0;
}


int main(int argc, char* argv[]) {
    if (argc != 4) {
        printf("Usage: %s <block-size> <file-name> <number-of-restarts>\n", argv[0]);
        exit(1);
    }
    // Collect input arguments
    int block_size = atoi(argv[1]);
    //printf("block size %d, \n", block_size);
    char* file_name = argv[2];
    int restarts = atoi(argv[3]);
    if(restarts <= 0){
        printf("Number of restarts has to be a number larger than 0");
        exit(1);
    }
    initHwd();

    // Collect information from datafile into distMatrix and cities
    uint32_t* distMatrix, *kerDist;
    distMatrix = (uint32_t*) malloc(sizeof(uint32_t) * MAXCITIES * MAXCITIES);
    int cities = fileToDistM(file_name, distMatrix);
    if( cities > MAXCITIES){
        printf("too many cities :( \n");
        exit(1);
    }
    distMatrix = (uint32_t*) realloc(distMatrix,sizeof(uint32_t)* cities * cities);
    hipMalloc((void**)&kerDist, cities*cities*sizeof(uint32_t));
    hipMemcpy(kerDist, distMatrix, cities*cities*sizeof(uint32_t), hipMemcpyHostToDevice);
    
    
    /*printf("cities: %d \n", cities );
    printf("matrix: \n");
    for (int i = 0; i < cities; i++){
        for (int j = 0; j < cities; j++){
            printf("%d ", distMatrix[i *cities + j]);
        }
        printf("\n");
    }
    printf("\n");*/


    //Calculate total number of iterations
    int totIter = ((cities-1) * (cities-2))/2;

    //Memory for i-array and j-array
    int *is_d, *js_d;
    hipMalloc((void**)&is_d, totIter*sizeof(uint32_t));
    hipMalloc((void**)&js_d, totIter*sizeof(uint32_t));


    init(block_size, cities, totIter, is_d, js_d);
    hipFree(js_d);

    /*int* is_h = (int*) malloc(totIter*sizeof(uint32_t));
    hipMemcpy(is_h, is_d, totIter*sizeof(uint32_t), hipMemcpyDeviceToHost);
    int k = 0;
    printf("is: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", is_h[i]);
        k++;
    }
    printf("]\n");
    printf("k = %d\n", k);

    int* js_h = (int*) malloc(totIter*sizeof(int));
    hipMemcpy(js_h, js_d, totIter*sizeof(int), hipMemcpyDeviceToHost);
    printf("js: [");
    for(int i = 0; i < totIter; i++){
        printf("%d, ", js_h[i]);
    }
    printf("]\n");
    free(js_h); free(is_h);*/
    

    //Create tour matrix row wise
    unsigned short *tourMatrixR_d; //*tourMatrixC_d;
    struct timeval randomTime;
    hipMalloc((void**)&tourMatrixR_d, (cities+1)*restarts*sizeof(unsigned short));
    unsigned int num_blocks_tour = (restarts + block_size-1)/block_size; 
    gettimeofday(&randomTime, NULL);
    int time = randomTime.tv_usec;
    createToursRowWise<<<num_blocks_tour, block_size>>> (tourMatrixR_d, cities, restarts, time);

    /*//Create tour matrix column wise
    hipMalloc((void**)&tourMatrixC_d, (cities+1)*restarts*sizeof(unsigned short));
    createToursColumnWise<<<num_blocks_tour, block_size>>> (tourMatrixC_d, cities, restarts);*/

    //run 2 opt kernel 
    //size_t sharedMemSize = (cities+1) * sizeof(unsigned short) + (block_size*3) * sizeof(int) + 3*sizeof(int);
    size_t sharedMemSize = (cities+1) * sizeof(unsigned short) + block_size * sizeof(ChangeTuple) + sizeof(ChangeTuple);
    //printf("sharedmemSize used in twoOptKer : %d \n", sharedMemSize);
    int *glo_results;
    hipMalloc((void**)&glo_results, 2*restarts*sizeof(int));
    twoOptKer2<<<restarts, block_size, sharedMemSize>>> (kerDist, tourMatrixR_d, 
                                                        is_d, glo_results, 
                                                        cities, totIter);
    //gpuErrchk( hipPeekAtLastError() );
 
    
    //run reduction of all local optimum cost across multiple blocks
    unsigned int num_blocks_gl_re = (num_blocks_tour+1)/2;
    size_t mult_sharedMem = (block_size*2) * sizeof(int);
    for(int i = num_blocks_gl_re; i > 1; i>>=1){
        multBlockReduce<<<i, block_size, mult_sharedMem>>>(glo_results, restarts);
        i++;
    }
    //run reduction on the last block
    multBlockReduce<<<1, block_size, mult_sharedMem>>>(glo_results, restarts);
    hipDeviceSynchronize();

    //print results
    int* glo_res = (int*) malloc(2*restarts*sizeof(int));
    hipMemcpy(glo_res, glo_results, 2*restarts*sizeof(int), hipMemcpyDeviceToHost);
    
    //tour matrix row wise
    unsigned short* tourMatrix_h = (unsigned short*) malloc((cities+1)*restarts*sizeof(unsigned short));
    hipMemcpy(tourMatrix_h, tourMatrixR_d, (cities+1)*restarts*sizeof(unsigned short), hipMemcpyDeviceToHost);

    
    /*//test tour matrix column wise
    unsigned short* tourMatrixC_h = (unsigned short*) malloc((cities+1)*restarts*sizeof(unsigned short));
    hipMemcpy(tourMatrixC_h, tourMatrixC_d, (cities+1)*restarts*sizeof(unsigned short), hipMemcpyDeviceToHost);
    
    printf("Tour R:  [");
    for(int i = 0; i < restarts; i++){
        printf("[");
        for(int j = 0; j < cities+1; j++){
                printf("%d, ", tourMatrix_h[i*(cities+1)+j]);
        }
        printf("]\n");
    }
    printf("]\n\n");

    printf("Tour C:  [");
    for(int i = 0; i < restarts; i++){
        printf("[");
        for(int j = 0; j < cities+1; j++){
                printf("%d, ", tourMatrixC_h[j*restarts+i]);
        }
        printf("]\n");
    }
    printf("]\n");
    free(tourMatrixC_h); hipFree(tourMatrixC_d);*/

    
    int tourId = glo_res[1];

    printf("Shortest path: %d\n", glo_res[0]);
    printf("Tour:  [");
    for(int i = 0; i < cities+1; i++){
        printf("%d, ", tourMatrix_h[(cities+1)*tourId+i]);
    }
    printf("]\n");


    free(distMatrix); free(tourMatrix_h); free(glo_res); 
    hipFree(is_d); hipFree(js_d); hipFree(tourMatrixR_d);
    hipFree(kerDist);
    hipFree(glo_results); 
    return 0;

    
}
