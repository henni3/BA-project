#include "tsp-main-helper.cu.h"

int main() {
    initHwd();

    //TEST: is i and j array correct?
    int block_size, cities, totIter, *is_d, *js_d, *is_h;
    block_size = 32;
    cities = 5;
    totIter = ((cities-1) * (cities-2))/2;

    hipMalloc((void**)&is_d, totIter*sizeof(uint32_t));
    hipMalloc((void**)&js_d, totIter*sizeof(uint32_t));
    is_h = (int*) malloc(totIter*sizeof(uint32_t));

    init(block_size, cities, totIter, is_d, js_d);

    hipMemcpy(is_h, is_d, totIter*sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("i     j\n")
    for(int ind = 0; ind < totIter; ind++){
        int num = is_h[ind];
        i = num >> 16;
        j = (num & 0xffff) + i + 2;
        printf("%d     %d\n", i,j);
    }
    printf("end\n")

    return 0;
}