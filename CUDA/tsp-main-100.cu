#include "tsp-main-helper.cu.h"

int main(int argc, char* argv[]) {
    if (argc != 4) {
        printf("Usage: %s <block-size> <file-name> <number-of-restarts>\n", argv[0]);
        exit(1);
    }
    // Collect input arguments
    int block_size = atoi(argv[1]);
    char* file_name = argv[2];
    int restarts = atoi(argv[3]);
    if(restarts <= 0){
        printf("Number of restarts has to be a number larger than 0");
        exit(1);
    }
    
    initHwd();

    //Create varibales
    struct timeval start, end, diff;
    uint32_t* distMatrix, *kerDist;
    int cities, totIter, *is_d, *js_d, *glo_results, *glo_res_h, tourId, elapsed;
    unsigned short *tourMatrixIn_d, *tourMatrixTrans_d, *tourMatrix_h;


    // Collect information from datafile into distMatrix and cities    
    distMatrix = (uint32_t*) malloc(sizeof(uint32_t) * MAXCITIES * MAXCITIES);
    cities = fileToDistM(file_name, distMatrix);
    if( cities > CITIES){
        printf("too many cities :( \n");
        exit(1);
    }
    distMatrix = (uint32_t*) realloc(distMatrix,sizeof(uint32_t) * cities * cities);
    hipMalloc((void**)&kerDist, cities*cities*sizeof(uint32_t));
    hipMemcpy(kerDist, distMatrix, cities*cities*sizeof(uint32_t), hipMemcpyHostToDevice);

    //Calculate total number of iterations
    totIter = ((cities-1) * (cities-2))/2;

    //Cuda malloc
    hipMalloc((void**)&tourMatrixIn_d, (cities+1)*restarts*sizeof(unsigned short));
    hipMalloc((void**)&tourMatrixTrans_d, (cities+1)*restarts*sizeof(unsigned short));
    hipMalloc((void**)&is_d, totIter*sizeof(uint32_t));
    hipMalloc((void**)&js_d, totIter*sizeof(uint32_t));
    hipMalloc((void**)&glo_results, 2*restarts*sizeof(int));

    //CPU malloc
    glo_res_h = (int*) malloc(2*sizeof(int));
    tourMatrix_h = (unsigned short*) malloc((cities+1)*restarts*sizeof(unsigned short));

    //Dry run program
    init(block_size, cities, totIter, is_d, js_d);
    run_kernels(tourMatrixIn_d, tourMatrixTrans_d, 
                is_d, kerDist, glo_results, 
                block_size, cities, restarts, totIter);
    
    hipDeviceSynchronize();
    //testing time for cities 100 program
    gettimeofday(&start, NULL); 
    for(int i = 0; i < GPU_RUNS; i++){
        //run program
        init(block_size, cities, totIter, is_d, js_d);
        run_kernels(tourMatrixIn_d, tourMatrixTrans_d, 
                    is_d, kerDist, glo_results, 
                    block_size, cities, restarts, totIter);

        //get results
        hipMemcpy(glo_res_h, glo_results, 2*sizeof(int), hipMemcpyDeviceToHost);
        tourId = glo_res_h[1];
    }

    hipDeviceSynchronize();
    gettimeofday(&end, NULL); 
    timeval_subtract(&diff, &end, &start);
    elapsed = (diff.tv_sec*1e6+diff.tv_usec) / GPU_RUNS; 
    printf("kernel 100 tour: Optimized Program runs on GPU in: %lu milisecs, repeats: %d\n", elapsed/1000, GPU_RUNS);
    
    //get results
    hipMemcpy(tourMatrix_h, tourMatrixTrans_d, (cities+1)*restarts*sizeof(unsigned short), hipMemcpyDeviceToHost);
    
    //print results
    printf("Shortest path: %d\n", glo_res_h[0]);
    printf("Tour:  [");
    for(int i = 0; i < cities+1; i++){
        printf("%d, ", tourMatrix_h[(cities+1)*tourId+i]);
    }
    printf("]\n");
    
    //Clean up
    free(distMatrix); free(tourMatrix_h); free(glo_res_h);  
    hipFree(is_d); hipFree(js_d); hipFree(tourMatrixTrans_d); hipFree(tourMatrixIn_d);
    hipFree(kerDist);
    hipFree(glo_results);
    return 0;
}
